#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <unistd.h>
//#include "nvToolsExt.h"

__global__ void saxpy(double *z, double *x, double *y, double alpha, int N) {

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = idx; i < N; i += stride) {
    //if (idx < N)
        z[i] = alpha*x[i] + y[i];
    }
}

int main(int argc, char *argv[]) {

    double *h_z, *h_x, *h_y;
    double *d_z, *d_x, *d_y;
    double alpha = 1.5;
    int N = 4096;
    int iterations = 2;

    MPI_Init(&argc, &argv);
    int commSize, commRank;
    MPI_Comm_size(MPI_COMM_WORLD, &commSize);
    MPI_Comm_rank(MPI_COMM_WORLD, &commRank);
	
    hipSetDevice(commRank);

    int c;
    char* endp;
    // parse arguments
    while ((c = getopt (argc, argv, "N:i:h")) != -1) {
        switch (c) {
	    case 'N':
	        N = strtol(optarg, &endp, 10);
		break;
	    case 'i':
	        iterations = strtol(optarg, &endp, 10);
	        break;
	    case  'h':
		printf("-N <problem_size> => default: -N 4096\n");
		printf("-i <number_of_iterations> => default: -i 2\n");
		exit(0);
	        break;
	    case '?':
		printf("Unknown argument. Use -h to see the options.\n");
		exit(1);
		break;
	}
    }
   
    if (commRank == 0) { 
        printf("Number of iterations: %d\n", iterations);
    	printf("Problem size (N): %d\n", N);
    }

    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Rank %d - Number of GPUs: %d\n", commRank, deviceCount);

    h_z = new double[N];
    h_x = new double[N];
    h_y = new double[N];
    
    for (int it = 0; it < iterations; it++){
	// initialize
	if (commRank == 0){
	    for (int i = 0; i < N; i += 1) {
                h_x[i] = 5.0;
                h_y[i] = -2.0;
		h_z[i] = 0.0;
	    }
	}
	
	// send the input arrays to the other process.
	if (commRank == 0) {
	    MPI_Send(h_x, N, MPI_DOUBLE, 1, it+0, MPI_COMM_WORLD);
	    MPI_Send(h_y, N, MPI_DOUBLE, 1, it+1, MPI_COMM_WORLD);
	}
	else if (commRank == 1) {
	    MPI_Recv(h_x, N, MPI_DOUBLE, 0, it+0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            MPI_Recv(h_y, N, MPI_DOUBLE, 0, it+1, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	}

	hipMalloc(&d_z, N*sizeof(double));
	hipMalloc(&d_y, N*sizeof(double));
	hipMalloc(&d_x, N*sizeof(double));

	// copy arrays from host to device
	hipMemcpy(d_x, h_x, N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, N*sizeof(double), hipMemcpyHostToDevice);
	
    	int threadsPerBlock = 512;
    	int numBlocks = 2; //N/threadsPerBlock + (N % threadsPerBlock != 0);

	// kernel call
	//nvtxRangePushA("saxpy");
    	saxpy<<<numBlocks, threadsPerBlock>>>(d_z, d_x, d_y, alpha, N);
    	//nvtxRangePop();

	// copy arrays back to the host
	hipMemcpy(h_z, d_z, N*sizeof(double), hipMemcpyDeviceToHost);

	// check if the results are correct
        bool success = true; 
	for (size_t i = 0; i < N; i += 1) {
            if (std::abs(h_z[i] - (1.5*5.0-2.0)) > 1E-8) {
                success = false;
	    }
        }
	if (!success) {
            printf("Rank %d => Error: incorrect results! it: %d\n", commRank, it);
	}
        else {
	    printf("Rank %d => Correct results! it: %d\n", commRank, it);
	}

	// send the result to rank 0.
	if (commRank == 1) {
	    MPI_Send(h_z, N, MPI_DOUBLE, 0, it+2, MPI_COMM_WORLD);
	}
	else if (commRank == 0) {
	    MPI_Recv(h_z, N, MPI_DOUBLE, 1, it+2, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	}
    }

    // cleaning
    delete[] h_x;
    delete[] h_y;
    delete[] h_z;	
    
    hipFree(d_z);
    hipFree(d_x);
    hipFree(d_y);

    MPI_Finalize();
    
    return 0;
}
